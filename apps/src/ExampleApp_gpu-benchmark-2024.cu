#include "hip/hip_runtime.h"
/*

Copyright (c) 2005-2023, University of Oxford.
All rights reserved.

University of Oxford means the Chancellor, Masters and Scholars of the
University of Oxford, having an administrative office at Wellington
Square, Oxford OX1 2JD, UK.

This file is part of Chaste.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
 * Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.
 * Neither the name of the University of Oxford nor the names of its
   contributors may be used to endorse or promote products derived from this
   software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE
GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT
OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

*/

/**
 * @file
 *
 * This file gives an example of how you can create your own executable
 * in a user project.
 */

#include <iostream>
#include <string>

#include "ExecutableSupport.hpp"
#include "Exception.hpp"
#include "PetscTools.hpp"
#include "PetscException.hpp"

#include "flamegpu/flamegpu.h"

#include "GPUModifier.cuh"
#include "NodesOnlyMesh.hpp"
#include "UniformCellCycleModel.hpp"
#include "OffLatticeSimulation.hpp"
#include "GeneralisedLinearSpringForce.hpp"
#include "CellsGenerator.hpp"
#include "TransitCellProliferativeType.hpp"
#include "SmartPointers.hpp"
#include "SimulationTime.hpp"

#include "Hello_gpu-benchmark-2024.hpp"

FLAMEGPU_AGENT_FUNCTION(test_do_nothing, flamegpu::MessageNone, flamegpu::MessageNone) {
    return flamegpu::ALIVE;
}

FLAMEGPU_INIT_FUNCTION(test_simple_force_create_agents) {
  // Retrieve the host agent tools for agent sheep in the default state
  flamegpu::HostAgentAPI cell = FLAMEGPU->agent("cell");

  // Create 10 new cell agents
  for (int i = 0; i < 3; ++i) {
      flamegpu::HostNewAgentAPI new_cell = cell.newAgent();
      new_cell.setVariable<float>("x", i * 0.5f);
      new_cell.setVariable<float>("y", i * 0.5f);
      new_cell.setVariable<float>("x_force", 0.0f);
      new_cell.setVariable<float>("y_force", 0.0f);
      new_cell.setVariable<float>("radius", 0.5f);
  }
}

FLAMEGPU_AGENT_FUNCTION(test_output_location, flamegpu::MessageNone, flamegpu::MessageBruteForce) {
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("radius", FLAMEGPU->getVariable<float>("radius"));
    return flamegpu::ALIVE;
}

// Models repulsion force without division/apoptosis
FLAMEGPU_AGENT_FUNCTION(test_compute_force_meineke_spring, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    const float x = FLAMEGPU->getVariable<float>("x");
    const float y = FLAMEGPU->getVariable<float>("y");
    float x_force = 0.0;
    float y_force = 0.0;
    float radius = FLAMEGPU->getVariable<float>("radius");

    for (const auto& message : FLAMEGPU->message_in) {
        float other_x = message.getVariable<float>("x");
        float other_y = message.getVariable<float>("y");
        float other_radius = message.getVariable<float>("radius");
        
        // Compute unit distance
        float x_dist = other_x - x;
        float y_dist = other_y - y;
        float distance_between_nodes = sqrt(x_dist * x_dist + y_dist * y_dist);

        float unit_x = x_dist / distance_between_nodes;
        float unit_y = y_dist / distance_between_nodes;
        
        // Only compute force if within cutoff distance and for positive distance
        const float cutoff_length = 1.5f;
        if (distance_between_nodes < cutoff_length && distance_between_nodes > 0.0f) {

            // Compute rest length
            const float rest_length = radius + other_radius; 
            const float rest_length_final = rest_length;
            
            // TODO: Should check here if newly divided or apoptosis happening


            // Compute the force
            float overlap = distance_between_nodes - rest_length;
            bool is_closer_than_rest_length = (overlap <= 0);
            const float spring_stiffness = 15.0f;
            const float multiplication_factor = 1.0f;

            
            // A reasonably stable simple force law
            if (is_closer_than_rest_length) //overlap is negative
            {
                //assert(overlap > -rest_length_final);
                x_force += multiplication_factor * spring_stiffness * unit_x * rest_length_final* log(1.0 + overlap/rest_length_final);
                y_force  = multiplication_factor * spring_stiffness * unit_y * rest_length_final* log(1.0 + overlap/rest_length_final);
            }
            else
            {
                double alpha = 5.0;
                x_force += multiplication_factor * spring_stiffness * unit_x * overlap * exp(-alpha * overlap/rest_length_final);
                y_force += multiplication_factor * spring_stiffness * unit_y * overlap * exp(-alpha * overlap/rest_length_final);
            }
        }

        
    }

    FLAMEGPU->setVariable<float>("x_force", x_force);        
    FLAMEGPU->setVariable<float>("y_force", y_force);        

    return flamegpu::ALIVE;
}

typedef struct ResultsRow {
    std::string type;
    double box_size;
    double run_time;
} ResultsRow;

void WriteResultsToFile(std::vector<ResultsRow> results, std::string fileName) {
    std::ofstream results_file(fileName);
    for (auto& row : results) {
        results_file << row.type << ", " << row.box_size << ", " << row.run_time << "\n";
    }
    std::cout << "Results written to " << fileName << "\n";
}

void PerformGPUSim(const double size_of_box, std::vector<ResultsRow>& results) {
    
    std::cout << "Starting GPU sim with box size: " << size_of_box << "\n";
    auto start_time = std::chrono::high_resolution_clock::now();
    
    SimulationTime::Instance()->SetStartTime(0.0);
    unsigned cells_across = size_of_box * 1.52;
    double scaling = size_of_box/(double(cells_across-1));

    // Create a simple 3D NodeBasedCellPopulation consisting of cells evenly spaced in a regular grid
    std::vector<Node<2>*> nodes;
    unsigned index = 0;
    for (unsigned i=0; i<cells_across; i++)
    {
        for (unsigned j=0; j<cells_across; j++)
        {
            nodes.push_back(new Node<2>(index, false,  (double) i * scaling , (double) j * scaling));
            index++;
        }
    }

    NodesOnlyMesh<2> mesh;
    mesh.ConstructNodesWithoutMesh(nodes, 1.5);

    std::vector<CellPtr> cells;
    MAKE_PTR(TransitCellProliferativeType, p_transit_type);
    CellsGenerator<UniformCellCycleModel, 2> cells_generator;
    cells_generator.GenerateBasicRandom(cells, mesh.GetNumNodes(), p_transit_type);

    NodeBasedCellPopulation<2> node_based_cell_population(mesh, cells);
    //node_based_cell_population.AddCellPopulationCountWriter<CellProliferativeTypesCountWriter>();

    // Set up cell-based simulation
    OffLatticeSimulation<2> simulator(node_based_cell_population);
    simulator.SetOutputDirectory("GPUNodeBased");
    simulator.SetSamplingTimestepMultiple(12);
    simulator.SetEndTime(1.0);

    MAKE_PTR(GPUModifier<2>, gpuModifier);
    simulator.AddSimulationModifier(gpuModifier);

    // Run simulation
    simulator.Solve();

    // Avoid memory leak
    for (unsigned i=0; i<nodes.size(); i++)
    {
        delete nodes[i];
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    
    ResultsRow row;
    row.type = "gpu";
    row.box_size = size_of_box;
    row.run_time = duration.count();
    results.push_back(row);
}

void PerformCPUSim(const double size_of_box, std::vector<ResultsRow>& results) {
    
    std::cout << "Starting CPU sim with box size: " << size_of_box << "\n";
    auto start_time = std::chrono::high_resolution_clock::now();
    SimulationTime::Instance()->SetStartTime(0.0);
    unsigned cells_across = size_of_box * 1.52;
    double scaling = size_of_box/(double(cells_across-1));

    // Create a simple 3D NodeBasedCellPopulation consisting of cells evenly spaced in a regular grid
    std::vector<Node<2>*> nodes;
    unsigned index = 0;
    for (unsigned i=0; i<cells_across; i++)
    {
        for (unsigned j=0; j<cells_across; j++)
        {
            nodes.push_back(new Node<2>(index, false,  (double) i * scaling , (double) j * scaling));
            index++;
        }
    }

    NodesOnlyMesh<2> mesh;
    mesh.ConstructNodesWithoutMesh(nodes, 1.5);

    std::vector<CellPtr> cells;
    MAKE_PTR(TransitCellProliferativeType, p_transit_type);
    CellsGenerator<UniformCellCycleModel, 2> cells_generator;
    cells_generator.GenerateBasicRandom(cells, mesh.GetNumNodes(), p_transit_type);

    NodeBasedCellPopulation<2> node_based_cell_population(mesh, cells);
    //node_based_cell_population.AddCellPopulationCountWriter<CellProliferativeTypesCountWriter>();

    // Set up cell-based simulation
    OffLatticeSimulation<2> simulator(node_based_cell_population);
    simulator.SetOutputDirectory("GPUNodeBased");
    simulator.SetSamplingTimestepMultiple(12);
    simulator.SetEndTime(1.0);

    MAKE_PTR(GeneralisedLinearSpringForce<2>, springForce);
    simulator.AddForce(springForce);

    // Run simulation
    simulator.Solve();

    // Avoid memory leak
    for (unsigned i=0; i<nodes.size(); i++)
    {
        delete nodes[i];
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    
    ResultsRow row;
    row.type = "cpu";
    row.box_size = size_of_box;
    row.run_time = duration.count();
    results.push_back(row);
}

int main(int argc, char *argv[])
{
    // This sets up PETSc and prints out copyright information, etc.
    ExecutableSupport::StandardStartup(&argc, &argv);
    ExecutableSupport::StandardStartup(&argc, &argv);
    std::vector<double> box_sizes = {10.0, 20.0, 30.0, 40.0, 50.0, 100.0, 200.0, 300.0, 400.0, 500.0};
    std::vector<ResultsRow> results;
    for (auto box_size : box_sizes) {
        PerformGPUSim(box_size, results);
        PerformCPUSim(box_size, results);
    }
    WriteResultsToFile(results, "results.txt");
    std::cout << "Benchmark complete\n";
}
